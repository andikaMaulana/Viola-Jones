
#include <hip/hip_runtime.h>
__device__
int rectanglesSum(int** integralImage, int x, int y, int w, int h)
{
    int A = x > 0 && y > 0 ? integralImage[x - 1][y - 1] : 0;
    int B = x + w > 0 && y > 0 ? integralImage[x + w - 1][y - 1] : 0;
    int C = x > 0 && y + h > 0 ? integralImage[x - 1][y + h - 1] : 0;
    int D = x + w > 0 && y + h > 0 ? integralImage[x + w - 1][y + h - 1] : 0;

    return A + D - B - C;
}

extern "C"
__global__ void haar_type_C(int** integralImage, int* allRectangles, int numRectangles, int* haarFeatures)
{
    // Get an "unique id" of the thread that correspond to one pixel
    const unsigned int tidX = blockIdx.x * blockDim.x + threadIdx.x;

    if (tidX < numRectangles)
    {

        int x = allRectangles[tidX * 4];
        int y = allRectangles[tidX * 4 + 1];
        int w = allRectangles[tidX * 4 + 2];
        int h = allRectangles[tidX * 4 + 3];

        int mid = h / 2;

        int r1 = rectanglesSum(integralImage, x, y, w, mid);

        int r2 = rectanglesSum(integralImage, x, y + mid, w, mid);

        haarFeatures[tidX] = r2 - r1;
    }

    __syncthreads();
}